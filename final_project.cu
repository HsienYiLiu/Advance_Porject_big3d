/*
Author: Hsien-Yi Liu
Subject: CSE 523.T52 Advanced Project in Computer Science I - Spring 2020
Data: May 20, 2020
*/

#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#define EXIT_FAILURE 1
#define X 0
#define Y 1
#define Z 2
#define MAX_INT   2147483647 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define DIM 3                  /* Dimension of points */
typedef int    tPointi[DIM];   /* Type integer point */
typedef double tPointd[DIM];   /* Type double point */
#define PMAX 1000000             /* Max # of pts */
typedef enum boolean{ FALSE, TRUE } boolean;
tPointd Vertices[PMAX];        /* All the points */
tPointi Faces[PMAX];           /* Each triangle face is 3 indices */
tPointd com_Vertices[PMAX];
tPointi com_Faces[PMAX];
int check = 0;
tPointi Box[PMAX][2];          /* Box around each face */
int n_facets, n_vertices;      /* Original polyhedron*/
int com_facets, com_vertices,counter;  /* Original polyhedron*/
int final[PMAX];
/*Function Define Table*/
void read_ori(void);
void read_com(void);
int ComputeBox( int F, tPointd bmin, tPointd bmax );
int irint( double x );
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box );
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax );
void RandomRay( tPointd ray, int radius );
void AddVec( tPointd q, tPointd ray );
__global__ void check_each( tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V,int F,tPointi * ori_F,tPointd * ori_V,tPointd * r,tPointd * q, tPointi *Box, int * out);
__global__ void check_segment(tPointd *ori_V, tPointi *ori_F, tPointd *q,int index,int * out);
/*GPU Error Check*/
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
int main(){
    time_t begin = time(NULL);
    int n, F, i,radius;
    tPointd q, bmin, bmax;
    // Load data from .off file
    read_ori();
    read_com();
    // Allocate memory
    n = n_vertices;
    F = n_facets;
    // Setting Boundary BOX
    for ( i = 0; i < DIM; i++ ){
        bmin[i] = bmax[i] = Vertices[0][i];
    }
    radius = ComputeBox( n, bmin, bmax );
    int counter = com_vertices - 1;
    tPointd *d_bmin, *d_bmax, *c_com_V,*ori_V,*final_r,*final_q;
    tPointi *cu_box,*ori_F;
    int *out,*result;
    //Allocate Memory
    result = (int *)malloc(sizeof(int)*F);
    gpuErrchk(hipMalloc(&c_com_V,sizeof(tPointd)*F));
    hipMalloc(&ori_V,sizeof(tPointd)*n);
    hipMalloc(&ori_F,sizeof(tPointi)*F);
    hipMalloc(&d_bmax,sizeof(tPointd)*3);
    hipMalloc(&d_bmin,sizeof(tPointd)*3);
    hipMalloc(&final_r,sizeof(tPointd)*3);
    hipMalloc(&final_q,sizeof(tPointd)*3); 
    hipMalloc(&cu_box,sizeof(tPointi)*2*F);
    hipMalloc(&out,sizeof(tPointi)*F);

    hipMemcpy(c_com_V, com_Vertices, sizeof(tPointd)*F, hipMemcpyHostToDevice);
    hipMemcpy(ori_V, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
    hipMemcpy(ori_F, Faces, sizeof(tPointi)*F, hipMemcpyHostToDevice);
    hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(cu_box, Box, sizeof(tPointi)*2*F, hipMemcpyHostToDevice);
    hipMemcpy(out, result, sizeof(int)*F, hipMemcpyHostToDevice);

    //Check whether the point is inside the polyhedron
    while( counter >= 0 ) {
        tPointd r;
        q[X] = com_Vertices[counter][X];
        q[Y] = com_Vertices[counter][Y];
        q[Z] = com_Vertices[counter][Z];
        printf( "\n %d -------->q = %lf %lf %lf\n", counter, q[X], q[Y], q[Z] );
        RandomRay( r, radius ); 
        AddVec( q, r ); // add the ray with the point to create end point
        hipMemcpy(final_q, q, sizeof(tPointd)*3, hipMemcpyHostToDevice);
        printf("Ray endpoint: (%lf,%lf,%lf)\n", r[0],r[1],r[2] );
        hipMemcpy(final_r, r, sizeof(tPointd)*3, hipMemcpyHostToDevice);
        check_each<<<F,1>>>(d_bmin,d_bmax,radius,c_com_V,F,ori_F, ori_V,final_r,final_q,cu_box, out);     
        hipMemcpy(result,out, sizeof(int)*F, hipMemcpyDeviceToHost);
        int total = 0;
        for(int i = 0; i < F; i++){
            total = total + result[i];
        }
        if(total % 2 != 1){
            break;
        }
        counter--;
    }
    // Check Segment
    free(result);
    hipFree(out);
    hipMalloc(&out,sizeof(tPointi)*F);
    hipMemcpy(out, result, sizeof(int)*F, hipMemcpyHostToDevice);
    result = (int *)malloc(sizeof(int)*F);
    int segment_check = 0;
    for(int i = 0; i < com_vertices; i++){
        check_segment<<<com_vertices,1>>>(ori_V,ori_F,c_com_V,i,out);
        hipMemcpy(result,out, sizeof(int)*F, hipMemcpyDeviceToHost);
        for(int i = 0; i < com_vertices; i++){
            if(result[i] != 0){
                segment_check = 1;
                break;
            }
        }
        if(segment_check == 1){
            printf("Inner polehedron isn't inside the outer polehedron\n");
            break;
        }
    }
    if(segment_check == 0){
        printf("Inner polehedron is inside the outer polehedron\n");
    }
    //printf("testt final %d\n", index);
    free(result);
    hipFree(d_bmin);hipFree(d_bmax);hipFree(c_com_V);
    hipFree(ori_F);hipFree(ori_V);hipFree(final_r);
    hipFree(final_q);hipFree(out);hipFree(cu_box);
    time_t end = time(NULL); 
    printf("Time elpased is %ld seconds \n", (end - begin));
    printf("Inner polyhedron fully contains in the outer polyhedron");
    return 0;
}
__device__ double Dot( tPointd a, tPointd b )
{
    int i;
    double sum = 0.0;
    for( i = 0; i < DIM; i++ )
       sum += a[i] * b[i];

    return  sum;
}
__device__ int PlaneCoeff(tPointd N)
{
    int i;
    double t;              /* Temp storage */
    double biggest = 0.0;  /* Largest component of normal vector. */
    int m = 0;             /* Index of largest component. */


    /* Find the largest component of N. */
    for ( i = 0; i < DIM; i++ ) {
      t = fabs( N[i] );
      if ( t > biggest ) {
        biggest = t;
        m = i;
      }
    }
    return m;
}
__device__ int SegPlaneInt(double D,double denom, double num, tPointd q, tPointd r)
{
    double t;
    
    //printf("SegPlaneInt: num=%lf, denom=%lf\n", q[0], q[1] );

    if ( denom == 0.0 ) {  /* Segment is parallel to plane. */
       if ( num == 0.0 )   /* q is on plane. */
           return 5;
       else
           return 0;
    }
    else
       t = num / denom;

    if ( (0.0 < t) && (t < 1.0) )
         //return '1';
         return 9;
    else if ( num == 0.0 )   //t == 0 
         return 6;
    else if ( num == denom ) //t == 1 
         return 7;
    else return 0;
}
__device__ int AreaSign( tPointd a, tPointd b, tPointd c )  
{
    double area2;

    area2 = ( b[0] - a[0] ) * ( c[1] - a[1] ) -
            ( c[0] - a[0] ) * ( b[1] - a[1] );

    /* The area should be an integer. */
    if      ( area2 >  0.5 ) return  1;
    else if ( area2 < -0.5 ) return -1;
    else                     return  0;
} 
__device__ int InTri2D( int area0, int area1, int area2 )
{
   /* compute three AreaSign() values for pp w.r.t. each edge of the face in 2D */

   if ( ( area0 == 0 ) && ( area1 > 0 ) && ( area2 > 0 ) ||
        ( area1 == 0 ) && ( area0 > 0 ) && ( area2 > 0 ) ||
        ( area2 == 0 ) && ( area0 > 0 ) && ( area1 > 0 ) ) 
     return 2;

   if ( ( area0 == 0 ) && ( area1 < 0 ) && ( area2 < 0 ) ||
        ( area1 == 0 ) && ( area0 < 0 ) && ( area2 < 0 ) ||
        ( area2 == 0 ) && ( area0 < 0 ) && ( area1 < 0 ) )
     return 2;                 
   
   if ( ( area0 >  0 ) && ( area1 > 0 ) && ( area2 > 0 ) ||
        ( area0 <  0 ) && ( area1 < 0 ) && ( area2 < 0 ) )
     return 3;

   if ( ( area0 == 0 ) && ( area1 == 0 ) && ( area2 == 0 ) )
     //printf( "Error in InTriD\n" ); exit(EXIT_FAILURE);
     return -3;    

   if ( ( area0 == 0 ) && ( area1 == 0 ) ||
        ( area0 == 0 ) && ( area2 == 0 ) ||
        ( area1 == 0 ) && ( area2 == 0 ) )
     return 1;

   else  
     return 4;  
}
__device__ int VolumeSign( tPointd a, tPointd b, tPointd c, tPointd d )
{ 
   double vol;
   double ax, ay, az, bx, by, bz, cx, cy, cz, dx, dy, dz;
   double bxdx, bydy, bzdz, cxdx, cydy, czdz;

   ax = a[X];
   ay = a[Y];
   az = a[Z];
   bx = b[X];
   by = b[Y];
   bz = b[Z];
   cx = c[X]; 
   cy = c[Y];
   cz = c[Z];
   dx = d[X];
   dy = d[Y];
   dz = d[Z];
   //printf("%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf \n",ax,ay,az,bx,by,bz,cx,cy,cz,dx);

   bxdx=bx-dx;
   bydy=by-dy;
   bzdz=bz-dz;
   cxdx=cx-dx;
   cydy=cy-dy;
   czdz=cz-dz;
   vol =   (az-dz) * (bxdx*cydy - bydy*cxdx)
         + (ay-dy) * (bzdz*cxdx - bxdx*czdz)
         + (ax-dx) * (bydy*czdz - bzdz*cydy);


   /* The volume should be an integer. */
   if      ( vol > 0.5 )   return  1;
   else if ( vol < -0.5 )  return -1;
   else                    return  0;
}
__device__ int SegTriCross(int vol0, int vol1, int vol2)
{
   
 
   //printf( "SegTriCross:  vol0 = %d; vol1 = %d; vol2 = %d\n", vol0, vol1, vol2 ); 
     
   /* Same sign: segment intersects interior of triangle. */
   if ( ( ( vol0 > 0 ) && ( vol1 > 0 ) && ( vol2 > 0 ) ) || 
        ( ( vol0 < 0 ) && ( vol1 < 0 ) && ( vol2 < 0 ) ) )
      return 33;
   
   /* Opposite sign: no intersection between segment and triangle */
   if ( ( ( vol0 > 0 ) || ( vol1 > 0 ) || ( vol2 > 0 ) ) &&
        ( ( vol0 < 0 ) || ( vol1 < 0 ) || ( vol2 < 0 ) ) )
      return 0;

   else if ( ( vol0 == 0 ) && ( vol1 == 0 ) && ( vol2 == 0 ) )
     //fprintf( stderr, "Error 1 in SegTriCross\n" ), exit(EXIT_FAILURE);
     return -3;   
 
   /* Two zeros: segment intersects vertex. */
   else if ( ( ( vol0 == 0 ) && ( vol1 == 0 ) ) || 
             ( ( vol0 == 0 ) && ( vol2 == 0 ) ) || 
             ( ( vol1 == 0 ) && ( vol2 == 0 ) ) )
      return 11;

   /* One zero: segment intersects edge. */
   else if ( ( vol0 == 0 ) || ( vol1 == 0 ) || ( vol2 == 0 ) )
      return 22;
   
   else
     return -3;
     //fprintf( stderr, "Error 2 in SegTriCross\n" ), exit(EXIT_FAILURE);
}
__global__ void check_segment(tPointd *ori_V, tPointi *ori_F, tPointd *q,int index, int *out){
      tPointd N,rq;
      int i = blockIdx.x;
      int j = i + 1;
      N[X] = (ori_V[ori_F[i][Z]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Y]][Y]-ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Y] = (ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Z])-(ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Z] = (ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Y]- ori_V[ori_F[i][X]][Y])*(ori_V[ori_F[i][Z]][X]- ori_V[ori_F[i][X]][X]);
      // Cal dot
      double D,num,denom;
      D = Dot( ori_V[ori_F[i][0]], N );
      
      int m = PlaneCoeff(N);
      num = D - Dot( *q, N );
      rq[X] = q[index][X] - q[j][X];
      rq[Y] = q[index][Y] - q[j][Y];
      rq[Z] = q[index][Z] - q[j][Z];
      denom = Dot(rq,N);
      out[i] = SegPlaneInt(D, denom, num, *q, *q);
      //printf("in check segment   %d, %d, %d\n",index, j, tmp_code);

}
__global__ void check_each( tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V,int F,tPointi * ori_F,tPointd * ori_V,tPointd * r,tPointd * q, tPointi *Box, int * out)
{
      
      volatile __shared__ bool FoundIt;
      // initialize shared status
      FoundIt = false;
      __syncthreads();
      int k = 0, crossings = 0;
      int code = -1; 
      //int i = blockIdx.x;
      int i = blockIdx.x;
      //printf("i %d \n",i);
      crossings = 0;
      // get N
      tPointd N,rq;
      N[X] = (ori_V[ori_F[i][Z]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Y]][Y]-ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Y] = (ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Z])-(ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Z] = (ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Y]- ori_V[ori_F[i][X]][Y])*(ori_V[ori_F[i][Z]][X]- ori_V[ori_F[i][X]][X]);
      // Cal dot
      double D,num,denom;
      D = Dot( ori_V[ori_F[i][0]], N );
      int m;
      m = PlaneCoeff(N);
      num = D - Dot( *q, N );
      rq[X] = r[0][X] - q[0][X];
      rq[Y] = r[0][Y] - q[0][Y];
      rq[Z] = r[0][Z] - q[0][Z];
      denom = Dot(rq,N);
      int tmp_code = SegPlaneInt(D, denom, num, *q, *r);

      if(i < F){
         if ( !InBox( *q, *bmin, *bmax ) == 2 ){
              out[i] = 0;
              FoundIt = true;
         }
         /*if (BoxTest( i, *q, *r, *Box ) == '0' && FoundIt == false) {
              
              out[i] = 0;
              FoundIt = true;
         }*/else 
         if(FoundIt == false){
             if(tmp_code == 8){
                 tmp_code = 8;
             }
             if(tmp_code == 6){
                 tPointd pp,Tp[3];     // projected T: three new vertices 
                 
                 // Project out coordinate m in both p and the triangular face 
                 int j = 0;
                 for ( i = 0; i < DIM; i++ ) {
                     if ( i != m ) {    //skip largest coordinate 
                         pp[j] = q[0][i];
                         for ( k = 0; k < 3; k++ ){
	                     Tp[k][j] = ori_V[ori_F[i][k]][i];
                             //printf(" plane=(%lf)\n", Tp[k][j]);
                         }
                         j++;
                          
                      }
                 }
                 int area0 = AreaSign( pp, Tp[0], Tp[1] );
                 int area1 = AreaSign( pp, Tp[1], Tp[2] );
                 int area2 = AreaSign( pp, Tp[2], Tp[0] );                 
                 tmp_code = InTri2D(  area0, area1, area2 );
             }
             else if(tmp_code == 7){
                 tPointd pp,Tp[3];     // projected T: three new vertices 
                 //t = num / denom;

                 // Project out coordinate m in both p and the triangular face 
                 int j = 0;
                 for ( i = 0; i < DIM; i++ ) {
                     if ( i != m ) {    //skip largest coordinate 
                         pp[j] = r[0][i];
                         for ( k = 0; k < 3; k++ ){
                             Tp[k][j] = ori_V[ori_F[i][k]][i];
                             //printf(" plane=(%lf)\n", Tp[k][j]);
                         }
                         j++;

                      }
                 }
                 int area0 = AreaSign( pp, Tp[0], Tp[1] );
                 int area1 = AreaSign( pp, Tp[1], Tp[2] );
                 int area2 = AreaSign( pp, Tp[2], Tp[0] );
                 tmp_code = InTri2D(  area0, area1, area2 );

             }else if(tmp_code == 9){
                 int vol0, vol1, vol2;
                 vol0 = VolumeSign( q[0], ori_V[ori_F[i][0] ], ori_V[ori_F[i][1] ], r[0] );
                 vol1 = VolumeSign( q[0], ori_V[ori_F[i][1] ], ori_V[ori_F[i][2] ], r[0] );
                 vol2 = VolumeSign( q[0], ori_V[ori_F[i][2] ], ori_V[ori_F[i][0] ], r[0] );
                 //printf( "SegTriCross:  vol0 = %d; vol1 = %d; vol2 = %d\n", vol0, vol1, vol2 ); 
                 tmp_code = SegTriCross(vol0,vol1,vol2);
                 //FoundIt = true;

             }else{
                 tmp_code = tmp_code;
         
             }
         }
         if(FoundIt == false){
           code = tmp_code;

           if( code == 5 || code == 11 || code == 22){
              printf("Degenerate ray\n");
              FoundIt = true;  
           }
         
           //If ray hits face at interior point, increment crossings.
           else if ( code == 33) {
              crossings++;
              printf( "crossings = %d\n", crossings );
           }

           //If query endpoint q sits on a V/E/F, return that code.
           else if ( code == 2 || code == 3|| code == 1)
              //return code;
              out[i] = code;

           //If ray misses triangle, do nothing. 
           else if ( code == 44 || code == 8|| code == 8 )
              ;

           else{
              out[i] = -3;
           }    
         }

         out[i] = crossings;
         
         }
}

__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax )
{
  //printf("baxxxx test %lf, %lf, %lf\n", q[Z],bmax[Z],q[X]);
  if( ( bmin[X] <= q[X] ) && ( q[X] <= bmax[X] ) &&
      ( bmin[Y] <= q[Y] ) && ( q[Y] <= bmax[Y] ) &&
      ( bmin[Z] <= q[Z] ) && ( q[Z] <= bmax[Z] ) ){
    //printf("TRUEEEE");
    return 1;
   
  }
  return 2;
}
/* Return a random ray endpoint */
 void RandomRay( tPointd ray, int radius )
{
  double x, y, z, w, t;
  /* Generate a random point on a sphere of radius 1. */
  /* the sphere is sliced at z, and a random point at angle t
     generated on the circle of intersection. */
  z = 2.0 * (double) rand() / MAX_INT - 1.0;
  t = 2.0 * M_PI * (double) rand() / MAX_INT;
  //printf("check %lf\n",rand1);
  w = sqrt( 1 - z*z );
  x = w * cos( t );
  y = w * sin( t );
  
  ray[X] = radius * x;
  ray[Y] = radius * y;
  ray[Z] = radius * z;
  
  /*printf( "RandomRay returns %6d %6d %6d\n", ray[X], ray[Y], ray[Z] );*/
}
void AddVec( tPointd q, tPointd ray )
{
  int i;
  
  for( i = 0; i < DIM; i++ )
    ray[i] = q[i] + ray[i];
}
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box)
{
   int i; /* Coordinate index */
   int w;
   //printf(" Box %d\n", Box[0][0][0]);
   for ( i=0; i < DIM; i++ ) {
       w = Box[n]; //min: lower left 
       if ( ((int)a[i] < w ) && ((int)b[i] < w) ) return '0';
       w = Box[n]; // max: upper right 
       if ( ((int)a[i] > w) && ((int)b[i] > w) ) return '0';
   }
   return '?';
}
__global__ void cal(tPointd *bmin, tPointd *bmax,tPointd *V,int F){

    int i = blockIdx.x; // will give you X block Index at that particular thread
    int j = blockIdx.y; // will give you Y block Index at that particular thread. 
    if(i < F){
        //j = j%3;
        for(j = 0; j < 3; j++){
            if( V[i][j] < *bmin[j] )
                *bmin[j] = V[i][j];
            if( V[i][j] > *bmax[j] ){
                *bmax[j] = V[i][j];
                //printf("V %lf\n",V[i][j]);
            }
 //           printf("Check i = %d, j = %d, F = %d\n",i,j,F);
        }
    }
   // printf("bmax %lf, bmin %lf \n",*bmax[Y],*bmin[Y]);
}
int ComputeBox( int n, tPointd bmin, tPointd bmax ){
  double radius;
  tPointd *min, *max,*d_bmax, *d_bmin, *d_a;
  min = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 
  max = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 

  hipMalloc(&d_a,sizeof(tPointd)*n);
  hipMalloc(&d_bmax,sizeof(tPointd)*3);
  hipMalloc(&d_bmin,sizeof(tPointd)*3);

  hipMemcpy(d_a, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
  hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);

  //dim3 blockSize(256);
  //dim3 gridSize((n + blockSize.x) / blockSize.x);
  cal<<<n+1, 1>>>(d_bmin, d_bmax, d_a, n);
  hipMemcpy(max,d_bmax, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  hipMemcpy(min,d_bmin, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  printf("------------------------\n");
  printf("bmax %lf bmin %lf \n",*max[X],*min[X]);
  printf("bmax %lf, bmin %lf \n",*max[Y],*min[Y]);
  printf("bmax %lf, bmin %lf \n",*max[Z],*min[Z]);
  radius = sqrt( pow( (double)(*max[X] - *min[X]), 2.0 ) +
                 pow( (double)(*max[Y] - *min[Y]), 2.0 ) +
                 pow( (double)(*max[Z] - *min[Z]), 2.0 ) );
  printf("radius = %lf\n", radius);
  bmax[0] = *max[0];bmax[1] = *max[1];bmax[2] = *max[2];
  bmin[0] = *min[0];bmin[1] = *min[1];bmin[2] = *min[2];
  
  //printf("radius = %lf\n", bmax[2]);
  hipFree(d_bmax);
  hipFree(d_bmin);
  hipFree(d_a);
  free(max);
  free(min);

  return irint( radius +1 ) + 1;
}

int irint( double x )
{
        return (int) rint( x );
}
void read_ori(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    //float a,b,c;
    fp = fopen("0.off", "r");
    int i = 0;
    int j,k,w;

    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL ) {
            // init facets and vertices
            if(count <= 2){
                if(token_count == 0){
                    n_vertices = atoi(token);
                }else if(token_count == 1){
                    n_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
                if(token_count == 0){
                    Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    Vertices[count - 4][Y] = atof(token);
                }else{
                    Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            } else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;

                if(token_count == 1){
                    Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    Faces[i][Y] = atoi(token);
                    //printf("->>>>  %d\n",Faces[count - 144][X]);
                }else if(token_count == 3){
                    Faces[i][Z] = atoi(token);
                    for ( j=0; j < 3; j++ ) {
                        Box[i][0][j] = Vertices[ Faces[i][0] ][j];
                        Box[i][1][j] = Vertices[ Faces[i][0] ][j];
                  }

               for ( k=1; k < 3; k++ )
               for ( j=0; j < 3; j++ ) {
                  w = Vertices[ Faces[i][k] ][j];
                  //printf("->>>>  %d\n",Faces[i][k]);
                  if ( w < Box[i][0][j] ) Box[i][0][j] = w;
                  if ( w > Box[i][1][j] ) Box[i][1][j] = w;
               }
               /*
               printf("Bounding box: (%d,%d,%d);(%d,%d,%d)\n",
                  Box[i][0][0],
                  Box[i][0][1],
                  Box[i][0][2],
                  Box[i][1][0],
                  Box[i][1][1],
                  Box[i][1][2] );
                */
                }
                token_count++;
            }
            token = strtok(NULL," ");
            //free(token);
        }
    }
    if (line)
        free(line);
}
void read_com(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    //float a,b,c;
    fp = fopen("demo.off", "r");
    int i ;
    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL) {
            // init facets and vertices
            if(count <= 2){
                printf("setting of file  %s\n", token);
                if(token_count == 0){
                    com_vertices = atoi(token);
                }else if(token_count == 1){
                    com_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
               if(token_count == 0){
                    com_Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    com_Vertices[count - 4][Y] = atof(token);
                }else{
                    com_Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            }else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;
                if(token_count == 1){
                    com_Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    com_Faces[i][Y] = atoi(token);
                }else if(token_count == 3){
                    com_Faces[i][Z] = atoi(token);
                }
                token_count++;
            }
            token = strtok(NULL, " ");
        }
    }
    if (line)
        free(line);
}
