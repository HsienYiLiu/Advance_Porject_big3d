#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#define EXIT_FAILURE 1
#define X 0
#define Y 1
#define Z 2
#define MAX_INT   2147483647 
//typedef enum { FALSE, TRUE } bool;

#define DIM 3                  /* Dimension of points */
typedef int    tPointi[DIM];   /* Type integer point */
typedef double tPointd[DIM];   /* Type double point */
#define PMAX 1000000             /* Max # of pts */
typedef enum boolean{ FALSE, TRUE } boolean;
tPointd Vertices[PMAX];        /* All the points */
tPointi Faces[PMAX];           /* Each triangle face is 3 indices */
tPointd com_Vertices[PMAX];
tPointi com_Faces[PMAX];
int check = 0;
tPointi Box[PMAX][2];          /* Box around each face */
int n_facets, n_vertices;      /* Original polyhedron*/
int com_facets, com_vertices;  /* Original polyhedron*/

void read_ori(void);
void read_com(void);
int ComputeBox( int F, tPointd bmin, tPointd bmax );
int irint( double x );
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box );
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax );
void RandomRay( tPointd ray, int radius );
void AddVec( tPointd q, tPointd ray );
int InPolyhedron( int F, tPointd q, tPointd bmin, tPointd bmax, int radius );
//__global__ void check_each( tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V,int F,tPointd * ori_F,tPointd * ori_V,tPointd * r,tPointd * q, int * out);
//read_ori();
int main(){
    int n, F, i;
    tPointd q, bmin, bmax;
    int radius;
    read_ori();
    read_com();
    n = n_vertices;
    F = n_facets;
    // Allocate memory
    for ( i = 0; i < DIM; i++ ){
        bmin[i] = bmax[i] = Vertices[0][i];
    }
    radius = ComputeBox( n, bmin, bmax );
    int counter = com_vertices - 1;
    while( counter >= 0 ) {
        q[X] = com_Vertices[counter][X];
        q[Y] = com_Vertices[counter][Y];
        q[Z] = com_Vertices[counter][Z];
        printf( "\n %d -------->q = %lf %lf %lf\n", counter, q[X], q[Y], q[Z] );
        printf( "In = %c\n", InPolyhedron( F, q, bmin, bmax, radius ) );
        counter--;
    }
    return 0;
}
__global__ void check_each( tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V,int F,tPointd * ori_F,tPointd * ori_V,tPointd * r,tPointd * q, tPointi *Box, int * out)
{
      
      volatile __shared__ bool FoundIt;
      // initialize shared status
      FoundIt = false;
      __syncthreads();
      int f, k = 0, crossings = 0;
      int code = -1;
      int i = blockIdx.x;
      crossings = 0;
      //f = &Box[0][0][0];
      if(i < F){
         //if ( !InBox( *q, *bmin, *bmax ) ){
         //     out[i] = 3;
         //     FoundIt = true;
         //     printf("wpwowow %d\n", out[i]);
         // }
         if ( BoxTest( f, *q, *r, *Box ) == '0' && FoundIt == false) {
              
              //printf("XXXXXXXXXXX \n");       
              out[i] = 4;
              //code = '0';
              printf("BoxTest = 0!\n");
         }
         else code = SegTriInt( Faces[f], q, r, p );
         printf( "Face = %d: BoxTest/SegTriInt returns %c\n\n", f, code );
         /* 
         //If ray is degenerate, then goto outer while to generate another.
         if ( code == 'p' || code == 'v' || code == 'e' ) {
            printf("Degenerate ray\n");
            goto LOOP;
         }
   
         //If ray hits face at interior point, increment crossings.
         else if ( code == 'f' ) {
            crossings++;
            printf( "crossings = %d\n", crossings );
         }

         //If query endpoint q sits on a V/E/F, return that code.
         else if ( code == 'V' || code == 'E' || code == 'F' )
            //return code;
            out[i] = code;

         //If ray misses triangle, do nothing. 
         else if ( code == '0' )
            ;

         else 
            fprintf( stderr, "Error, exit(EXIT_FAILURE)\n" ), exit(1);

       */
       }
}

int InPolyhedron( int F, tPointd q, tPointd bmin, tPointd bmax, int radius )
{
    tPointd r,p;  /* Intersection point; not used. */
    int f, k = 0, crossings = 0;
    tPointd *d_bmin, *d_bmax, *c_com_V,*ori_F,*ori_V,*final_r,*final_q;
    tPointi *cu_box;
    int *out,*result;
    printf("pppppp\n");
    //char result[counter];
    result = (int *)malloc(sizeof(int)*F);
    
   
    hipMalloc(&c_com_V,sizeof(tPointd)*F);
    hipMalloc(&ori_V,sizeof(tPointd)*F);
    hipMalloc(&ori_F,sizeof(tPointd)*F);
    hipMalloc(&d_bmax,sizeof(tPointd)*3);
    hipMalloc(&d_bmin,sizeof(tPointd)*3);
    hipMalloc(&final_r,sizeof(tPointd)*3);
    hipMalloc(&final_q,sizeof(tPointd)*3); 
    hipMalloc(&cu_box,sizeof(tPointi)*2*F);

    hipMemcpy(c_com_V, com_Vertices, sizeof(tPointd)*F, hipMemcpyHostToDevice);
    hipMemcpy(ori_V, Vertices, sizeof(tPointd)*F, hipMemcpyHostToDevice);
    hipMemcpy(ori_F, Faces, sizeof(tPointd)*F, hipMemcpyHostToDevice);
    hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(final_q, q, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(cu_box, Box, sizeof(tPointi)*2*F, hipMemcpyHostToDevice);
    hipMemcpy(out, result, sizeof(int)*F, hipMemcpyHostToDevice);

    //printf("Box test %d\n",cu_box[0][0][0]);
   
   //LOOP:
   //while( k++ < F) {
      crossings = 0;
  
      RandomRay( r, radius ); 
      AddVec( q, r ); // add the ray with the point to create end point
      
      printf("Ray endpoint: (%lf,%lf,%lf)\n", r[0],r[1],r[2] );
      hipMemcpy(final_r, r, sizeof(tPointd)*3, hipMemcpyHostToDevice);
      check_each<<<F, 1>>>(d_bmin,d_bmax,radius,c_com_V,F,ori_F, ori_V,final_r,final_q,cu_box, out);     
      hipMemcpy(result,out, sizeof(int)*F, hipMemcpyDeviceToHost);
      
     // break;

   //}  
   /*printf( "Crossings = %d\n", crossings );
   // q strictly interior to polyhedron iff an odd number of crossings.
   if( ( crossings % 2 ) == 1 )
      //return   'i';
      out[i] = 1;
   //else return 'o';
   else out[i] = 9;
   */
   //printf("result -->  %d\n", result[i]);
   free(result);
   hipFree(d_bmin);hipFree(d_bmax);hipFree(c_com_V);
   hipFree(ori_F);hipFree(ori_V);hipFree(final_r);
   hipFree(final_q);hipFree(out);hipFree(cu_box);
   return 0;
}
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax )
{
  int i;

  if( ( bmin[X] <= q[X] ) && ( q[X] <= bmax[X] ) &&
      ( bmin[Y] <= q[Y] ) && ( q[Y] <= bmax[Y] ) &&
      ( bmin[Z] <= q[Z] ) && ( q[Z] <= bmax[Z] ) )
    return TRUE;
  return FALSE;
}
/* Return a random ray endpoint */
 void RandomRay( tPointd ray, int radius )
{
  double x, y, z, w, t;
  /*int tId = threadIdx.x + (blockIdx.x * blockDim.x);
  hiprandState state;
  hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

  double rand1 = hiprand_uniform_double(&state);
  double rand2 = hiprand_uniform_double(&state);*/
  /* Generate a random point on a sphere of radius 1. */
  /* the sphere is sliced at z, and a random point at angle t
     generated on the circle of intersection. */
  z = 2.0 * (double) rand() / MAX_INT - 1.0;
  t = 2.0 * M_PI * (double) rand() / MAX_INT;
  //printf("check %lf\n",rand1);
  w = sqrt( 1 - z*z );
  x = w * cos( t );
  y = w * sin( t );
  
  ray[X] = radius * x;
  ray[Y] = radius * y;
  ray[Z] = radius * z;
  
  /*printf( "RandomRay returns %6d %6d %6d\n", ray[X], ray[Y], ray[Z] );*/
}
void AddVec( tPointd q, tPointd ray )
{
  int i;
  
  for( i = 0; i < DIM; i++ )
    ray[i] = q[i] + ray[i];
}
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box)
{
   int i; /* Coordinate index */
   int w;
   //printf(" Box %d\n", w);
   for ( i=0; i < DIM; i++ ) {
       w = Box[n]; //min: lower left 
       if ( ((int)a[i] < w ) && ((int)b[i] < w) ) return '0';
       w = Box[n]; // max: upper right 
       if ( ((int)a[i] > w) && ((int)b[i] > w) ) return '0';
   }
   return '?';
}
__global__ void cal(tPointd *bmin, tPointd *bmax,tPointd *V,int F){

    int i = blockIdx.x; // will give you X block Index at that particular thread
    int j = blockIdx.y; // will give you Y block Index at that particular thread. 
    if(i < F){
        //j = j%3;
        for(j = 0; j < 3; j++){
            if( V[i][j] < *bmin[j] )
                *bmin[j] = V[i][j];
            if( V[i][j] > *bmax[j] ){
                *bmax[j] = V[i][j];
                //printf("V %lf\n",V[i][j]);
            }
            printf("Check i = %d, j = %d, F = %d\n",i,j,F);
        }
    }
    printf("bmax %lf, bmin %lf \n",*bmax[Y],*bmin[Y]);
}
int ComputeBox( int n, tPointd bmin, tPointd bmax ){
  int i, j;
  double radius;
  tPointd *d_bmin, *d_bmax, *d_a, *max, *min;
  max = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array1 on host
  min = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 

  hipMalloc(&d_a,sizeof(tPointd)*n);
  hipMalloc(&d_bmax,sizeof(tPointd)*3);
  hipMalloc(&d_bmin,sizeof(tPointd)*3);

  hipMemcpy(d_a, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
  hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);

  //dim3 blockSize(256);
  //dim3 gridSize((n + blockSize.x) / blockSize.x);
  cal<<<n, 1>>>(d_bmin, d_bmax, d_a, n);
  hipMemcpy(max,d_bmax, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  hipMemcpy(min,d_bmin, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  printf("------------------------\n");
  printf("bmax %lf bmin %lf \n",*max[X],*min[X]);
  printf("bmax %lf, bmin %lf \n",*max[Y],*min[Y]);
  printf("bmax %lf, bmin %lf \n",*max[Z],*min[Z]);
  radius = sqrt( pow( (double)(*max[X] - *min[X]), 2.0 ) +
                 pow( (double)(*max[Y] - *min[Y]), 2.0 ) +
                 pow( (double)(*max[Z] - *min[Z]), 2.0 ) );
  printf("radius = %lf\n", radius);
  hipFree(d_bmax);
  hipFree(d_bmin);
  hipFree(d_a);
  free(max);
  free(min);

  return irint( radius +1 ) + 1;
}

int irint( double x )
{
        return (int) rint( x );
}
void read_ori(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("big.off", "r");
    int i = 0;
    int j,k,n,w;

    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL ) {
            // init facets and vertices
            if(count <= 2){
                if(token_count == 0){
                    n_vertices = atoi(token);
                }else if(token_count == 1){
                    n_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
                if(token_count == 0){
                    Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    Vertices[count - 4][Y] = atof(token);
                }else{
                    Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            } else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;

                if(token_count == 1){
                    Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    Faces[i][Y] = atoi(token);
                    //printf("->>>>  %d\n",Faces[count - 144][X]);
                }else if(token_count == 3){
                    Faces[i][Z] = atoi(token);
                    for ( j=0; j < 3; j++ ) {
                        Box[i][0][j] = Vertices[ Faces[i][0] ][j];
                        Box[i][1][j] = Vertices[ Faces[i][0] ][j];
                  }

               for ( k=1; k < 3; k++ )
               for ( j=0; j < 3; j++ ) {
                  w = Vertices[ Faces[i][k] ][j];
                  //printf("->>>>  %d\n",Faces[i][k]);
                  if ( w < Box[i][0][j] ) Box[i][0][j] = w;
                  if ( w > Box[i][1][j] ) Box[i][1][j] = w;
               }
               /*
               printf("Bounding box: (%d,%d,%d);(%d,%d,%d)\n",
                  Box[i][0][0],
                  Box[i][0][1],
                  Box[i][0][2],
                  Box[i][1][0],
                  Box[i][1][1],
                  Box[i][1][2] );
                */
                }
                token_count++;
            }
            token = strtok(NULL," ");
            //free(token);
        }
    }
    if (line)
        free(line);
}
void read_com(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("t.off", "r");
    int i ;
    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL) {
            // init facets and vertices
            if(count <= 2){
                printf("setting of file  %s\n", token);
                if(token_count == 0){
                    com_vertices = atoi(token);
                }else if(token_count == 1){
                    com_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
               if(token_count == 0){
                    com_Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    com_Vertices[count - 4][Y] = atof(token);
                }else{
                    com_Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            }else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;
                if(token_count == 1){
                    com_Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    com_Faces[i][Y] = atoi(token);
                }else if(token_count == 3){
                    com_Faces[i][Z] = atoi(token);
                }
                token_count++;
            }
            token = strtok(NULL, " ");
        }
    }
    if (line)
        free(line);
}
