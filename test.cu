
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#define EXIT_FAILURE 1
#define X 0
#define Y 1
#define Z 2
#define MAX_INT   2147483647 
//typedef enum { FALSE, TRUE } bool;

#define DIM 3                  /* Dimension of points */
typedef int    tPointi[DIM];   /* Type integer point */
typedef double tPointd[DIM];   /* Type double point */
#define PMAX 1000000             /* Max # of pts */
typedef enum boolean{ FALSE, TRUE } boolean;
tPointd Vertices[PMAX];        /* All the points */
tPointi Faces[PMAX];           /* Each triangle face is 3 indices */
tPointd com_Vertices[PMAX];
tPointi com_Faces[PMAX];
int check = 0;
tPointi Box[PMAX][2];          /* Box around each face */
int n_facets, n_vertices;      /* Original polyhedron*/
int com_facets, com_vertices;  /* Original polyhedron*/

void read_ori(void);
void read_com(void);
int ComputeBox( int F, tPointd bmin, tPointd bmax );
int irint( double x );
char BoxTest ( int n, tPointd a, tPointd b );
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax );
__device__ void RandomRay( tPointd ray, int radius );
__device__ void AddVec( tPointd q, tPointd ray );
__global__ void InPolyhedron( int F,  tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V, int * out);
//read_ori();
int main(){
    int n, F, i;
    tPointd q, bmin, bmax;
    int radius;
    read_ori();
    read_com();
    n = n_vertices;
    F = n_facets;
    // Allocate memory
    for ( i = 0; i < DIM; i++ ){
        bmin[i] = bmax[i] = Vertices[0][i];
    }
    radius = ComputeBox( n, bmin, bmax );
    int counter = com_vertices - 1;
    //printf("counter %d\n,", counter);
    // setting for judge
    tPointd *d_bmin, *d_bmax, *c_com_V;
    int *out,*result;
    
    //char out[counter];
    //char result[counter];
    result = (int *)malloc(sizeof(int)*counter);
   
    hipMalloc(&c_com_V,sizeof(tPointd)*counter);
    hipMalloc(&d_bmax,sizeof(tPointd)*3);
    hipMalloc(&d_bmin,sizeof(tPointd)*3);
    hipMalloc(&out,sizeof(int)*counter);

    hipMemcpy(c_com_V, com_Vertices, sizeof(tPointd)*counter, hipMemcpyHostToDevice);
    hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(out, result, sizeof(int)*counter, hipMemcpyHostToDevice);
    InPolyhedron<<<counter, 1>>>(n,d_bmin, d_bmax,radius,c_com_V, out);
    printf("111111\n");
    hipMemcpy(result,out, sizeof(int)*counter, hipMemcpyDeviceToHost);
    printf("2222222\n");
    printf("resultFFFF %d\n",result[1]);

    hipFree(d_bmax);
    hipFree(d_bmin);
    hipFree(c_com_V);
    hipFree(out);
    free(result);   

    //InPolyhedron( F, q, bmin, bmax, radius )
    /*
    while( counter >= 0 ) {
        q[X] = com_Vertices[counter][X];
        q[Y] = com_Vertices[counter][Y];
        q[Z] = com_Vertices[counter][Z];
        printf( "\n %d -------->q = %lf %lf %lf\n", counter, q[X], q[Y], q[Z] );
        //printf( "In = %c\n", InPolyhedron( F, q, bmin, bmax, radius ) );
        counter--;
    }*/
    return 0;
}
__global__ void check_each(int F,tPointd * ori_F,tPointd * r, tPointd *q){

}

__global__ void InPolyhedron( int F,  tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V, int * out )
{
   //volatile bool *found = FALSE;
   volatile __shared__ bool FoundIt;
   // initialize shared status
    FoundIt = false;
   //__syncthreads();
   tPointd r;  /* Rayendpoint. */
   tPointd p,q;  /* Intersection point; not used. */
   int f, k = 0, crossings = 0;
   int code = -1;
   int i = blockIdx.x;
   printf("i %d\n",i);
   q[0] = c_com_V[i][0];
   q[1] = c_com_V[i][1];
   q[2] = c_com_V[i][2];
   /* If query point is outside bounding box, finished. */
   if ( !InBox( q, *bmin, *bmax ) ){
      out[i] = 3;
      FoundIt = true;
      printf("wpwowow %d\n", out[i]);
   }
      //return 'o';
   
   LOOP:
   while( k++ < F && FoundIt == false) {
      crossings = 0;
  
      RandomRay( r, radius ); 
      AddVec( q, r ); // add the ray with the point to create end point
      printf("Ray endpoint: (%lf,%lf,%lf)\n", r[0],r[1],r[2] );
      /*
      for ( f = 0; f < F; f++ ) {  // Begin check each face 
         if ( BoxTest( f, q, r ) == '0' ) {
              out[i] = '0';
              //code = '0';
              printf("BoxTest = 0!\n");
         }
         //else code = SegTriInt( Faces[f], q, r, p );
         printf( "Face = %d: BoxTest/SegTriInt returns %c\n\n", f, code );

         //If ray is degenerate, then goto outer while to generate another.
         if ( code == 'p' || code == 'v' || code == 'e' ) {
            printf("Degenerate ray\n");
            goto LOOP;
         }
   
         //If ray hits face at interior point, increment crossings.
         else if ( code == 'f' ) {
            crossings++;
            printf( "crossings = %d\n", crossings );
         }

         //If query endpoint q sits on a V/E/F, return that code.
         else if ( code == 'V' || code == 'E' || code == 'F' )
            //return code;
            out[i] = code;

         //If ray misses triangle, do nothing. 
         else if ( code == '0' )
            ;

         else 
            fprintf( stderr, "Error, exit(EXIT_FAILURE)\n" ), exit(1);

      } */
      break;

   }  
   printf( "Crossings = %d\n", crossings );
   /* q strictly interior to polyhedron iff an odd number of crossings. */
   if( ( crossings % 2 ) == 1 )
      //return   'i';
      out[i] = 1;
   //else return 'o';
   else out[i] = 9;
   free(r);
   printf("result -->  %d\n", out[i]);
}
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax )
{
  int i;

  if( ( bmin[X] <= q[X] ) && ( q[X] <= bmax[X] ) &&
      ( bmin[Y] <= q[Y] ) && ( q[Y] <= bmax[Y] ) &&
      ( bmin[Z] <= q[Z] ) && ( q[Z] <= bmax[Z] ) )
    return TRUE;
  return FALSE;
}
/* Return a random ray endpoint */
__device__ void RandomRay( tPointd ray, int radius )
{
  double x, y, z, w, t;
  int tId = threadIdx.x + (blockIdx.x * blockDim.x);
  hiprandState state;
  hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

  double rand1 = hiprand_uniform_double(&state);
  double rand2 = hiprand_uniform_double(&state);
  /* Generate a random point on a sphere of radius 1. */
  /* the sphere is sliced at z, and a random point at angle t
     generated on the circle of intersection. */
  z = 2.0 * (double) 0 / MAX_INT - 1.0;
  t = 2.0 * M_PI * (double) rand2 / MAX_INT;
  printf("check %lf\n",rand1);
  w = sqrt( 1 - z*z );
  x = w * cos( t );
  y = w * sin( t );
  
  ray[X] = radius * x;
  ray[Y] = radius * y;
  ray[Z] = radius * z;
  
  /*printf( "RandomRay returns %6d %6d %6d\n", ray[X], ray[Y], ray[Z] );*/
}
__device__ void AddVec( tPointd q, tPointd ray )
{
  int i;
  
  for( i = 0; i < DIM; i++ )
    ray[i] = q[i] + ray[i];
}
char BoxTest ( int n, tPointd a, tPointd b )
{
   int i; /* Coordinate index */
   int w;

   for ( i=0; i < DIM; i++ ) {
       w = Box[ n ][0][i]; /* min: lower left */
       if ( ((int)a[i] < w ) && ((int)b[i] < w) ) return '0';
       w = Box[ n ][1][i]; /* max: upper right */
       if ( ((int)a[i] > w) && ((int)b[i] > w) ) return '0';
   }
   return '?';
}
__global__ void cal(tPointd *bmin, tPointd *bmax,tPointd *V,int F){

    int i = blockIdx.x; // will give you X block Index at that particular thread
    int j = blockIdx.y; // will give you Y block Index at that particular thread. 
    if(i < F){
        //j = j%3;
        for(j = 0; j < 3; j++){
            if( V[i][j] < *bmin[j] )
                *bmin[j] = V[i][j];
            if( V[i][j] > *bmax[j] ){
                *bmax[j] = V[i][j];
                //printf("V %lf\n",V[i][j]);
            }
            printf("Check i = %d, j = %d, F = %d\n",i,j,F);
        }
    }
    printf("bmax %lf, bmin %lf \n",*bmax[Y],*bmin[Y]);
}
int ComputeBox( int n, tPointd bmin, tPointd bmax ){
  int i, j;
  double radius;
  tPointd *d_bmin, *d_bmax, *d_a, *max, *min;
  max = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array1 on host
  min = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 

  hipMalloc(&d_a,sizeof(tPointd)*n);
  hipMalloc(&d_bmax,sizeof(tPointd)*3);
  hipMalloc(&d_bmin,sizeof(tPointd)*3);

  hipMemcpy(d_a, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
  hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);

  //dim3 blockSize(256);
  //dim3 gridSize((n + blockSize.x) / blockSize.x);
  cal<<<n, 1>>>(d_bmin, d_bmax, d_a, n);
  hipMemcpy(max,d_bmax, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  hipMemcpy(min,d_bmin, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  printf("------------------------\n");
  printf("bmax %lf bmin %lf \n",*max[X],*min[X]);
  printf("bmax %lf, bmin %lf \n",*max[Y],*min[Y]);
  printf("bmax %lf, bmin %lf \n",*max[Z],*min[Z]);
  radius = sqrt( pow( (double)(*max[X] - *min[X]), 2.0 ) +
                 pow( (double)(*max[Y] - *min[Y]), 2.0 ) +
                 pow( (double)(*max[Z] - *min[Z]), 2.0 ) );
  printf("radius = %lf\n", radius);
  hipFree(d_bmax);
  hipFree(d_bmin);
  hipFree(d_a);
  free(max);
  free(min);

  return irint( radius +1 ) + 1;
}

int irint( double x )
{
        return (int) rint( x );
}
void read_ori(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("big.off", "r");
    int i = 0;
    int j,k,n,w;

    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL ) {
            // init facets and vertices
            if(count <= 2){
                if(token_count == 0){
                    n_vertices = atoi(token);
                }else if(token_count == 1){
                    n_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
                if(token_count == 0){
                    Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    Vertices[count - 4][Y] = atof(token);
                }else{
                    Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            } else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;

                if(token_count == 1){
                    Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    Faces[i][Y] = atoi(token);
                    //printf("->>>>  %d\n",Faces[count - 144][X]);
                }else if(token_count == 3){
                    Faces[i][Z] = atoi(token);
                    for ( j=0; j < 3; j++ ) {
                        Box[i][0][j] = Vertices[ Faces[i][0] ][j];
                        Box[i][1][j] = Vertices[ Faces[i][0] ][j];
                  }

               for ( k=1; k < 3; k++ )
               for ( j=0; j < 3; j++ ) {
                  w = Vertices[ Faces[i][k] ][j];
                  //printf("->>>>  %d\n",Faces[i][k]);
                  if ( w < Box[i][0][j] ) Box[i][0][j] = w;
                  if ( w > Box[i][1][j] ) Box[i][1][j] = w;
               }
               /*
               printf("Bounding box: (%d,%d,%d);(%d,%d,%d)\n",
                  Box[i][0][0],
                  Box[i][0][1],
                  Box[i][0][2],
                  Box[i][1][0],
                  Box[i][1][1],
                  Box[i][1][2] );
                */
                }
                token_count++;
            }
            token = strtok(NULL," ");
            //free(token);
        }
    }
    if (line)
        free(line);
}
void read_com(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("t.off", "r");
    int i ;
    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL) {
            // init facets and vertices
            if(count <= 2){
                printf("setting of file  %s\n", token);
                if(token_count == 0){
                    com_vertices = atoi(token);
                }else if(token_count == 1){
                    com_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
               if(token_count == 0){
                    com_Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    com_Vertices[count - 4][Y] = atof(token);
                }else{
                    com_Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            }else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;
                if(token_count == 1){
                    com_Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    com_Faces[i][Y] = atoi(token);
                }else if(token_count == 3){
                    com_Faces[i][Z] = atoi(token);
                }
                token_count++;
            }
            token = strtok(NULL, " ");
        }
    }
    if (line)
        free(line);
}
