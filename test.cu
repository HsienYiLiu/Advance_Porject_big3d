
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#define EXIT_FAILURE 1
#define X 0
#define Y 1
#define Z 2
#define MAX_INT   2147483647 
//typedef enum { FALSE, TRUE } bool;

#define DIM 3                  /* Dimension of points */
typedef int    tPointi[DIM];   /* Type integer point */
typedef double tPointd[DIM];   /* Type double point */
#define PMAX 1000000             /* Max # of pts */
typedef enum boolean{ FALSE, TRUE } boolean;
tPointd Vertices[PMAX];        /* All the points */
tPointi Faces[PMAX];           /* Each triangle face is 3 indices */
tPointd com_Vertices[PMAX];
tPointi com_Faces[PMAX];
int check = 0;
tPointi Box[PMAX][2];          /* Box around each face */
int n_facets, n_vertices;      /* Original polyhedron*/
int com_facets, com_vertices;  /* Original polyhedron*/

void read_ori(void);
void read_com(void);
int ComputeBox( int F, tPointd bmin, tPointd bmax );
int irint( double x ); 
//read_ori();
int main(){
    int n, F, i;
    tPointd q, bmin, bmax;
    int radius;
    read_ori();
    read_com();
    n = n_vertices;
    F = n_facets;
    // Allocate memory
    for ( i = 0; i < DIM; i++ ){
        bmin[i] = bmax[i] = Vertices[0][i];
    }
    radius = ComputeBox( n, bmin, bmax );
    int counter = com_vertices - 1;   
    printf("counter %d\n,", counter);
    while( counter >= 0 ) {
        q[X] = com_Vertices[counter][X];
        q[Y] = com_Vertices[counter][Y];
        q[Z] = com_Vertices[counter][Z];
        printf( "\n %d -------->q = %lf %lf %lf\n", counter, q[X], q[Y], q[Z] );
        //printf( "In = %c\n", InPolyhedron( F, q, bmin, bmax, radius ) );
        counter--;
    }
    return 0;
}
__global__ void cal(tPointd *bmin, tPointd *bmax,tPointd *V,int F){
    
    int i = blockIdx.x; // will give you X block Index at that particular thread
    int j = blockIdx.y; // will give you Y block Index at that particular thread. 
    if(i < F){
        //j = j%3;
        for(j = 0; j < 3; j++){
            if( V[i][j] < *bmin[j] )
                *bmin[j] = V[i][j];
            if( V[i][j] > *bmax[j] ){
                *bmax[j] = V[i][j];
                //printf("V %lf\n",V[i][j]);
            }
            printf("Check i = %d, j = %d, F = %d\n",i,j,F);
        }
    }
    printf("bmax %lf, bmin %lf \n",*bmax[Y],*bmin[Y]);
}
int ComputeBox( int n, tPointd bmin, tPointd bmax ){ 
  int i, j;
  double radius;
  tPointd *d_bmin, *d_bmax, *d_a, *max, *min;
  max = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array1 on host
  min = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 
  
  hipMalloc(&d_a,sizeof(tPointd)*n); 
  hipMalloc(&d_bmax,sizeof(tPointd)*3);
  hipMalloc(&d_bmin,sizeof(tPointd)*3);
  
  hipMemcpy(d_a, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice); 
  hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice); 
  hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice); 
  /*
  for( i = 0; i < n; i++ )
    for( j = 0; j < DIM; j++ ) {
      if( Vertices[i][j] < bmin[j] )
              bmin[j] = Vertices[i][j];
      if( Vertices[i][j] > bmax[j] ) 
              bmax[j] = Vertices[i][j];
    }
  */
  dim3 blockSize(256);
  dim3 gridSize((n + blockSize.x) / blockSize.x);
  cal<<<n, 1>>>(d_bmin, d_bmax, d_a, n);
  hipMemcpy(max,d_bmax, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost); 
  hipMemcpy(min,d_bmin, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost); 
  printf("------------------------\n");
  printf("bmax %lf bmin %lf \n",*max[X],*min[X]);
  printf("bmax %lf, bmin %lf \n",*max[Y],*min[Y]);
  printf("bmax %lf, bmin %lf \n",*max[Z],*min[Z]);
  radius = sqrt( pow( (double)(*max[X] - *min[X]), 2.0 ) +
                 pow( (double)(*max[Y] - *min[Y]), 2.0 ) +
                 pow( (double)(*max[Z] - *min[Z]), 2.0 ) );
  printf("radius = %lf\n", radius);
  hipFree(d_bmax);
  hipFree(d_bmin);
  hipFree(d_a);
  free(max);
  free(min);
  //free(bmin);
  //free(bmax);
  printf("test");
  /*
  radius = sqrt( pow( (double)(bmax[X] - bmin[X]), 2.0 ) +
                 pow( (double)(bmax[Y] - bmin[Y]), 2.0 ) +
                 pow( (double)(bmax[Z] - bmin[Z]), 2.0 ) );
  printf("radius = %lf\n", radius);*/
  return irint( radius +1 ) + 1;
}

int irint( double x )
{       
        return (int) rint( x );
}
void read_ori(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("big.off", "r");
    int i = 0;
    int j,k,n,w;
    
    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " "); 
        int token_count = 0;
        while (token != NULL ) {  
            // init facets and vertices
            if(count <= 2){
                if(token_count == 0){
                    n_vertices = atoi(token);
                }else if(token_count == 1){
                    n_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
                if(token_count == 0){
                    Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    Vertices[count - 4][Y] = atof(token);
                }else{
                    Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            } else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;
                
                if(token_count == 1){
                    Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    Faces[i][Y] = atoi(token);
                    //printf("->>>>  %d\n",Faces[count - 144][X]);
                }else if(token_count == 3){
                    Faces[i][Z] = atoi(token);
                    for ( j=0; j < 3; j++ ) {
                        Box[i][0][j] = Vertices[ Faces[i][0] ][j];
                        Box[i][1][j] = Vertices[ Faces[i][0] ][j];  
                  }
               
               for ( k=1; k < 3; k++ )
               for ( j=0; j < 3; j++ ) {
                  w = Vertices[ Faces[i][k] ][j];
                  //printf("->>>>  %d\n",Faces[i][k]);
                  if ( w < Box[i][0][j] ) Box[i][0][j] = w;
                  if ( w > Box[i][1][j] ) Box[i][1][j] = w;
               }
               
               printf("Bounding box: (%d,%d,%d);(%d,%d,%d)\n",
                  Box[i][0][0],
                  Box[i][0][1],
                  Box[i][0][2],
                  Box[i][1][0],
                  Box[i][1][1],
                  Box[i][1][2] );
               
                }
                token_count++;
            }
            token = strtok(NULL," "); 
            //free(token);
        } 
    }
    if (line)
        free(line);
}
void read_com(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("t.off", "r");
    int i ;
    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " "); 
        int token_count = 0;
        while (token != NULL) {  
            // init facets and vertices
            if(count <= 2){
                printf("setting of file  %s\n", token);
                if(token_count == 0){
                    com_vertices = atoi(token);
                }else if(token_count == 1){
                    com_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
                if(token_count == 0){
                    com_Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    com_Vertices[count - 4][Y] = atof(token);
                }else{
                    com_Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            }else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;
                if(token_count == 1){
                    com_Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    com_Faces[i][Y] = atoi(token);
                }else if(token_count == 3){
                    com_Faces[i][Z] = atoi(token);
                }
                token_count++;
            } 
            token = strtok(NULL, " "); 
        } 
    }
    if (line)
        free(line);
}
