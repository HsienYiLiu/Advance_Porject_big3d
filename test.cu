
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#define EXIT_FAILURE 1
#define X 0
#define Y 1
#define Z 2
#define MAX_INT   2147483647 
//typedef enum { FALSE, TRUE } bool;

#define DIM 3                  /* Dimension of points */
typedef int    tPointi[DIM];   /* Type integer point */
typedef double tPointd[DIM];   /* Type double point */
#define PMAX 1000000             /* Max # of pts */
typedef enum boolean{ FALSE, TRUE } boolean;
tPointd Vertices[PMAX];        /* All the points */
tPointi Faces[PMAX];           /* Each triangle face is 3 indices */
tPointd com_Vertices[PMAX];
tPointi com_Faces[PMAX];
int check = 0;
tPointi Box[PMAX][2];          /* Box around each face */
int n_facets, n_vertices;      /* Original polyhedron*/
int com_facets, com_vertices;  /* Original polyhedron*/

void read_ori(void);
void read_com(void);
int ComputeBox( int F, tPointd bmin, tPointd bmax );
int irint( double x );
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box );
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax );
void RandomRay( tPointd ray, int radius );
void AddVec( tPointd q, tPointd ray );
int InPolyhedron( int F,int n, tPointd q, tPointd bmin, tPointd bmax, int radius );
//__global__ void check_each( tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V,int F,tPointd * ori_F,tPointd * ori_V,tPointd * r,tPointd * q, int * out);
//read_ori();
int main(){
    int n, F, i;
    tPointd q, bmin, bmax;
    int radius;
    read_ori();
    read_com();
    n = n_vertices;
    F = n_facets;
    // Allocate memory
    for ( i = 0; i < DIM; i++ ){
        bmin[i] = bmax[i] = Vertices[0][i];
    }
    radius = ComputeBox( n, bmin, bmax );
    int counter = com_vertices - 1;
    while( counter >= 0 ) {
        q[X] = com_Vertices[counter][X];
        q[Y] = com_Vertices[counter][Y];
        q[Z] = com_Vertices[counter][Z];
        printf( "\n %d -------->q = %lf %lf %lf\n", counter, q[X], q[Y], q[Z] );
        printf( "In = %c\n", InPolyhedron( F,n, q, bmin, bmax, radius ) );
        counter--;
    }
    return 0;
}
__device__ double Dot( tPointd a, tPointd b )
{
    int i;
    double sum = 0.0;
    for( i = 0; i < DIM; i++ )
       sum += a[i] * b[i];

    return  sum;
}
__device__ int PlaneCoeff(tPointd N)
{
    int i;
    double t;              /* Temp storage */
    double biggest = 0.0;  /* Largest component of normal vector. */
    int m = 0;             /* Index of largest component. */


    /* Find the largest component of N. */
    for ( i = 0; i < DIM; i++ ) {
      t = fabs( N[i] );
      if ( t > biggest ) {
        biggest = t;
        m = i;
      }
    }
    return m;
}
__device__ int SegPlaneInt(double D,double denom, double num, tPointd q, tPointd r)
{
    int i;
    double t;
    
    //printf("SegPlaneInt: num=%lf, denom=%lf\n", q[0], q[1] );

    if ( denom == 0.0 ) {  /* Segment is parallel to plane. */
       if ( num == 0.0 )   /* q is on plane. */
           return 10;
       else
           return 0;
    }
    else
       t = num / denom;
    //printf("SegPlaneInt: t=%lf \n", t );
    
    /*for( i = 0; i < DIM; i++ ){
       p[i] = q[i] + t * ( r[i] - q[i] );
    }*/

    if ( (0.0 < t) && (t < 1.0) )
         //return '1';
         return 1;
    else if ( num == 0.0 )   //t == 0 
         return 8;
    else if ( num == denom ) //t == 1 
         return 7;
    else return 0;
}
__device__ int AreaSign( tPointd a, tPointd b, tPointd c )  
{
    double area2;

    area2 = ( b[0] - a[0] ) * ( c[1] - a[1] ) -
            ( c[0] - a[0] ) * ( b[1] - a[1] );

    /* The area should be an integer. */
    if      ( area2 >  0.5 ) return  1;
    else if ( area2 < -0.5 ) return -1;
    else                     return  0;
} 
__device__ int InTri2D( int area0, int area1, int area2 )
{
   /* compute three AreaSign() values for pp w.r.t. each edge of the face in 2D */

   if ( ( area0 == 0 ) && ( area1 > 0 ) && ( area2 > 0 ) ||
        ( area1 == 0 ) && ( area0 > 0 ) && ( area2 > 0 ) ||
        ( area2 == 0 ) && ( area0 > 0 ) && ( area1 > 0 ) ) 
     return 3;

   if ( ( area0 == 0 ) && ( area1 < 0 ) && ( area2 < 0 ) ||
        ( area1 == 0 ) && ( area0 < 0 ) && ( area2 < 0 ) ||
        ( area2 == 0 ) && ( area0 < 0 ) && ( area1 < 0 ) )
     return 3;                 
   
   if ( ( area0 >  0 ) && ( area1 > 0 ) && ( area2 > 0 ) ||
        ( area0 <  0 ) && ( area1 < 0 ) && ( area2 < 0 ) )
     return 4;

   if ( ( area0 == 0 ) && ( area1 == 0 ) && ( area2 == 0 ) )
     //printf( "Error in InTriD\n" ); exit(EXIT_FAILURE);
     return 0;    

   if ( ( area0 == 0 ) && ( area1 == 0 ) ||
        ( area0 == 0 ) && ( area2 == 0 ) ||
        ( area1 == 0 ) && ( area2 == 0 ) )
     return 2;

   else  
     return 0;  
}
__device__ int VolumeSign( tPointd a, tPointd b, tPointd c, tPointd d )
{ 
   double vol;
   double ax, ay, az, bx, by, bz, cx, cy, cz, dx, dy, dz;
   double bxdx, bydy, bzdz, cxdx, cydy, czdz;

   ax = a[X];
   ay = a[Y];
   az = a[Z];
   bx = b[X];
   by = b[Y];
   bz = b[Z];
   cx = c[X]; 
   cy = c[Y];
   cz = c[Z];
   dx = d[X];
   dy = d[Y];
   dz = d[Z];
   //printf("%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf \n",ax,ay,az,bx,by,bz,cx,cy,cz,dx);

   bxdx=bx-dx;
   bydy=by-dy;
   bzdz=bz-dz;
   cxdx=cx-dx;
   cydy=cy-dy;
   czdz=cz-dz;
   vol =   (az-dz) * (bxdx*cydy - bydy*cxdx)
         + (ay-dy) * (bzdz*cxdx - bxdx*czdz)
         + (ax-dx) * (bydy*czdz - bzdz*cydy);


   /* The volume should be an integer. */
   if      ( vol > 0.5 )   return  1;
   else if ( vol < -0.5 )  return -1;
   else                    return  0;
}
__device__ int SegTriCross(int vol0, int vol1, int vol2)
{
   
 
   //printf( "SegTriCross:  vol0 = %d; vol1 = %d; vol2 = %d\n", vol0, vol1, vol2 ); 
     
   /* Same sign: segment intersects interior of triangle. */
   if ( ( ( vol0 > 0 ) && ( vol1 > 0 ) && ( vol2 > 0 ) ) || 
        ( ( vol0 < 0 ) && ( vol1 < 0 ) && ( vol2 < 0 ) ) )
      return 4;
   
   /* Opposite sign: no intersection between segment and triangle */
   if ( ( ( vol0 > 0 ) || ( vol1 > 0 ) || ( vol2 > 0 ) ) &&
        ( ( vol0 < 0 ) || ( vol1 < 0 ) || ( vol2 < 0 ) ) )
      return 0;

   else if ( ( vol0 == 0 ) && ( vol1 == 0 ) && ( vol2 == 0 ) )
     //fprintf( stderr, "Error 1 in SegTriCross\n" ), exit(EXIT_FAILURE);
     return -1;   
 
   /* Two zeros: segment intersects vertex. */
   else if ( ( ( vol0 == 0 ) && ( vol1 == 0 ) ) || 
             ( ( vol0 == 0 ) && ( vol2 == 0 ) ) || 
             ( ( vol1 == 0 ) && ( vol2 == 0 ) ) )
      return 2;

   /* One zero: segment intersects edge. */
   else if ( ( vol0 == 0 ) || ( vol1 == 0 ) || ( vol2 == 0 ) )
      return 3;
   
   else
     return -1;
     //fprintf( stderr, "Error 2 in SegTriCross\n" ), exit(EXIT_FAILURE);
}
__global__ void check_each( tPointd * bmin, tPointd * bmax,int radius, tPointd * c_com_V,int F,tPointi * ori_F,tPointd * ori_V,tPointd * r,tPointd * q, tPointi *Box, int * out)
{
      
      volatile __shared__ bool FoundIt;
      // initialize shared status
      FoundIt = false;
      __syncthreads();
      int f, k = 0, crossings = 0;
      int code = -1; 
      int i = blockIdx.x;
      crossings = 0;
      // get N
      tPointd N,rq;
      N[X] = (ori_V[ori_F[i][Z]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Y]][Y]-ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Y] = (ori_V[ori_F[i][Y]][Z]- ori_V[ori_F[i][X]][Z])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Z])-(ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y]);
      N[Z] = (ori_V[ori_F[i][Y]][X]- ori_V[ori_F[i][X]][X])*(ori_V[ori_F[i][Z]][Y]- ori_V[ori_F[i][X]][Y])-(ori_V[ori_F[i][Y]][Y]- ori_V[ori_F[i][X]][Y])*(ori_V[ori_F[i][Z]][X]- ori_V[ori_F[i][X]][X]);
      // Cal dot
      double D,num,denom,t;
      D = Dot( ori_V[ori_F[i][0]], N );
      int m;
      m = PlaneCoeff(N);
      num = D - Dot( *q, N );
      rq[X] = r[0][X] - q[0][X];
      rq[Y] = r[0][Y] - q[0][Y];
      rq[Z] = r[0][Z] - q[0][Z];
      denom = Dot(rq,N);
      int tmp_code = SegPlaneInt(D, denom, num, *q, *r);
      t = num / denom;

      //printf("SegPlaneInt: %d\n", tmp_code );      
      //f = &Box[0][0][0];
      //tmp_code = 1;
      if(i < F){
         if ( !InBox( *q, *bmin, *bmax ) ){
              out[i] = 0;
              FoundIt = true;
              //printf("wpwowow %d\n", out[i]);
         }
         if (BoxTest( f, *q, *r, *Box ) == '0' && FoundIt == false) {
              
              out[i] = 0;
              FoundIt = true;
              //printf("BoxTest = 0!\n");
         }
         else if(FoundIt == false){
             if(tmp_code == 0){
                 tmp_code = 0;
                 //FoundIt == true;
             }
             if(tmp_code == 8){
                 tPointd pp,Tp[3];     // projected T: three new vertices 
                 //t = num / denom;
                 
                 // Project out coordinate m in both p and the triangular face 
                 int j = 0;
                 for ( i = 0; i < DIM; i++ ) {
                     if ( i != m ) {    //skip largest coordinate 
                         pp[j] = q[0][i];
                         for ( k = 0; k < 3; k++ ){
	                     Tp[k][j] = ori_V[ori_F[i][k]][i];
                             //printf(" plane=(%lf)\n", Tp[k][j]);
                         }
                         j++;
                          
                      }
                 }
                 int area0 = AreaSign( pp, Tp[0], Tp[1] );
                 int area1 = AreaSign( pp, Tp[1], Tp[2] );
                 int area2 = AreaSign( pp, Tp[2], Tp[0] );
                 tmp_code = InTri2D(  area0, area1, area2 );
                 //FoundIt == true;
                 //printf("areaaa %d\n", out[i]);
                 //code = InTri2D( Tp, pp );
             }
             else if(tmp_code == 7){
                 tPointd pp,Tp[3];     // projected T: three new vertices 
                 //t = num / denom;

                 // Project out coordinate m in both p and the triangular face 
                 int j = 0;
                 for ( i = 0; i < DIM; i++ ) {
                     if ( i != m ) {    //skip largest coordinate 
                         pp[j] = r[0][i];
                         for ( k = 0; k < 3; k++ ){
                             Tp[k][j] = ori_V[ori_F[i][k]][i];
                             //printf(" plane=(%lf)\n", Tp[k][j]);
                         }
                         j++;

                      }
                 }
                 int area0 = AreaSign( pp, Tp[0], Tp[1] );
                 int area1 = AreaSign( pp, Tp[1], Tp[2] );
                 int area2 = AreaSign( pp, Tp[2], Tp[0] );
                 tmp_code = InTri2D(  area0, area1, area2 );
                 //FoundIt == true;
                 //printf("areaaa %d\n", out[i]);
                 //code = InTri2D( Tp, pp );
             //}else if(tmp_code == 10){
                 //out[i] = 10;
                 //FoundIt == true;
             }else if(tmp_code == 1){
                 int vol0, vol1, vol2;
                 vol0 = VolumeSign( q[0], ori_V[ori_F[i][0] ], ori_V[ori_F[i][1] ], r[0] );
                 vol1 = VolumeSign( q[0], ori_V[ori_F[i][1] ], ori_V[ori_F[i][2] ], r[0] );
                 vol2 = VolumeSign( q[0], ori_V[ori_F[i][2] ], ori_V[ori_F[i][0] ], r[0] );
                 //printf( "SegTriCross:  vol0 = %d; vol1 = %d; vol2 = %d\n", vol0, vol1, vol2 ); 
                 out[i] = SegTriCross(vol0,vol1,vol2);
                 //FoundIt = true;

             }else{
                 tmp_code = tmp_code;
         
             }
         }
         code = tmp_code;
         //code = 10;
         //printf( "Face = %d: BoxTest/SegTriInt returns %c\n\n", i, code ); 
          
         //If ray is degenerate, then goto outer while to generate another.
         if ( code == 10 || code == 2 || code == 3 ) {
            printf("Degenerate ray\n");
            out[i] = -3;
            FoundIt = true;  
            //printf("out %d\n",out[i]);
         }
         
         //If ray hits face at interior point, increment crossings.
         else if ( code == 4 ) {
            crossings++;
            printf( "crossings = %d\n", crossings );
         }

         //If query endpoint q sits on a V/E/F, return that code.
         else if ( code == 2 || code == 3|| code == 4)
            //return code;
            out[i] = code;

         //If ray misses triangle, do nothing. 
         else if ( code == 0 )
            ;

         else{
            out[i] = -1;
         } 
            //fprintf( stderr, "Error, exit(EXIT_FAILURE)\n" ), exit(1);      
       }
       if( ( crossings % 2 ) == 1 )
          out[i] = 1;
       else out[i] = 0;
       printf("check if every point is check i -> %d, out -> %d \n",i,out[i]);
}

int InPolyhedron( int F,int n, tPointd q, tPointd bmin, tPointd bmax, int radius )
{
    tPointd r,p;  /* Intersection point; not used. */
    int f, k = 0, crossings = 0;
    tPointd *d_bmin, *d_bmax, *c_com_V,*ori_V,*final_r,*final_q;
    tPointi *cu_box,*ori_F;
    int *out,*result;
    //char result[counter];
    result = (int *)malloc(sizeof(int)*F);
    
   
    hipMalloc(&c_com_V,sizeof(tPointd)*F);
    hipMalloc(&ori_V,sizeof(tPointd)*n);
    hipMalloc(&ori_F,sizeof(tPointi)*F);
    hipMalloc(&d_bmax,sizeof(tPointd)*3);
    hipMalloc(&d_bmin,sizeof(tPointd)*3);
    hipMalloc(&final_r,sizeof(tPointd));
    hipMalloc(&final_q,sizeof(tPointd)); 
    hipMalloc(&cu_box,sizeof(tPointi)*2*F);
    hipMalloc(&out,sizeof(tPointi)*F);

    hipMemcpy(c_com_V, com_Vertices, sizeof(tPointd)*F, hipMemcpyHostToDevice);
    hipMemcpy(ori_V, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
    hipMemcpy(ori_F, Faces, sizeof(tPointi)*F, hipMemcpyHostToDevice);
    hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
    hipMemcpy(final_q, q, sizeof(tPointd), hipMemcpyHostToDevice);
    hipMemcpy(cu_box, Box, sizeof(tPointi)*2*F, hipMemcpyHostToDevice);
    hipMemcpy(out, result, sizeof(int)*F, hipMemcpyHostToDevice);

    //printf("Box test %d\n",cu_box[0][0][0]);
   
   //LOOP:
    //while( k++ < F) {
      crossings = 0;
  
      RandomRay( r, radius ); 
      AddVec( q, r ); // add the ray with the point to create end point
      
      printf("Ray endpoint: (%lf,%lf,%lf)\n", r[0],r[1],r[2] );
      hipMemcpy(final_r, r, sizeof(tPointd), hipMemcpyHostToDevice);
      check_each<<<F, 1>>>(d_bmin,d_bmax,radius,c_com_V,F,ori_F, ori_V,final_r,final_q,cu_box, out);     
      hipMemcpy(result,out, sizeof(int)*F, hipMemcpyDeviceToHost);
      //printf("RRResult %d\n",result[k]);   
     // break;

   //}  
   /*printf( "Crossings = %d\n", crossings );
   // q strictly interior to polyhedron iff an odd number of crossings.
   if( ( crossings % 2 ) == 1 )
      //return   'i';
      out[i] = 1;
   //else return 'o';
   else out[i] = 9;
   */
   //printf("result -->  %d\n", result[i]);
   free(result);
   hipFree(d_bmin);hipFree(d_bmax);hipFree(c_com_V);
   hipFree(ori_F);hipFree(ori_V);hipFree(final_r);
   hipFree(final_q);hipFree(out);hipFree(cu_box);
   return 0;
}
__device__ int InBox( tPointd q, tPointd bmin, tPointd bmax )
{
  int i;

  if( ( bmin[X] <= q[X] ) && ( q[X] <= bmax[X] ) &&
      ( bmin[Y] <= q[Y] ) && ( q[Y] <= bmax[Y] ) &&
      ( bmin[Z] <= q[Z] ) && ( q[Z] <= bmax[Z] ) )
    return TRUE;
  return FALSE;
}
/* Return a random ray endpoint */
 void RandomRay( tPointd ray, int radius )
{
  double x, y, z, w, t;
  /* Generate a random point on a sphere of radius 1. */
  /* the sphere is sliced at z, and a random point at angle t
     generated on the circle of intersection. */
  z = 2.0 * (double) rand() / MAX_INT - 1.0;
  t = 2.0 * M_PI * (double) rand() / MAX_INT;
  //printf("check %lf\n",rand1);
  w = sqrt( 1 - z*z );
  x = w * cos( t );
  y = w * sin( t );
  
  ray[X] = radius * x;
  ray[Y] = radius * y;
  ray[Z] = radius * z;
  
  /*printf( "RandomRay returns %6d %6d %6d\n", ray[X], ray[Y], ray[Z] );*/
}
void AddVec( tPointd q, tPointd ray )
{
  int i;
  
  for( i = 0; i < DIM; i++ )
    ray[i] = q[i] + ray[i];
}
__device__ char BoxTest ( int n, tPointd a, tPointd b, tPointi Box)
{
   int i; /* Coordinate index */
   int w;
   //printf(" Box %d\n", w);
   for ( i=0; i < DIM; i++ ) {
       w = Box[n]; //min: lower left 
       if ( ((int)a[i] < w ) && ((int)b[i] < w) ) return '0';
       w = Box[n]; // max: upper right 
       if ( ((int)a[i] > w) && ((int)b[i] > w) ) return '0';
   }
   return '?';
}
__global__ void cal(tPointd *bmin, tPointd *bmax,tPointd *V,int F){

    int i = blockIdx.x; // will give you X block Index at that particular thread
    int j = blockIdx.y; // will give you Y block Index at that particular thread. 
    if(i < F){
        //j = j%3;
        for(j = 0; j < 3; j++){
            if( V[i][j] < *bmin[j] )
                *bmin[j] = V[i][j];
            if( V[i][j] > *bmax[j] ){
                *bmax[j] = V[i][j];
                //printf("V %lf\n",V[i][j]);
            }
 //           printf("Check i = %d, j = %d, F = %d\n",i,j,F);
        }
    }
   // printf("bmax %lf, bmin %lf \n",*bmax[Y],*bmin[Y]);
}
int ComputeBox( int n, tPointd bmin, tPointd bmax ){
  int i, j;
  double radius;
  tPointd *d_bmin, *d_bmax, *d_a, *max, *min;
  max = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array1 on host
  min = (tPointd *)malloc(sizeof(tPointd)*DIM); // Allocate array2 on host 

  hipMalloc(&d_a,sizeof(tPointd)*n);
  hipMalloc(&d_bmax,sizeof(tPointd)*3);
  hipMalloc(&d_bmin,sizeof(tPointd)*3);

  hipMemcpy(d_a, Vertices, sizeof(tPointd)*n, hipMemcpyHostToDevice);
  hipMemcpy(d_bmin, bmin, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);
  hipMemcpy(d_bmax, bmax, sizeof(tPointd)*DIM, hipMemcpyHostToDevice);

  //dim3 blockSize(256);
  //dim3 gridSize((n + blockSize.x) / blockSize.x);
  cal<<<n, 1>>>(d_bmin, d_bmax, d_a, n);
  hipMemcpy(max,d_bmax, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  hipMemcpy(min,d_bmin, sizeof(tPointd)*DIM, hipMemcpyDeviceToHost);
  //printf("------------------------\n");
  //printf("bmax %lf bmin %lf \n",*max[X],*min[X]);
  //printf("bmax %lf, bmin %lf \n",*max[Y],*min[Y]);
  //printf("bmax %lf, bmin %lf \n",*max[Z],*min[Z]);
  radius = sqrt( pow( (double)(*max[X] - *min[X]), 2.0 ) +
                 pow( (double)(*max[Y] - *min[Y]), 2.0 ) +
                 pow( (double)(*max[Z] - *min[Z]), 2.0 ) );
  printf("radius = %lf\n", radius);
  hipFree(d_bmax);
  hipFree(d_bmin);
  hipFree(d_a);
  free(max);
  free(min);

  return irint( radius +1 ) + 1;
}

int irint( double x )
{
        return (int) rint( x );
}
void read_ori(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("big.off", "r");
    int i = 0;
    int j,k,n,w;

    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL ) {
            // init facets and vertices
            if(count <= 2){
                if(token_count == 0){
                    n_vertices = atoi(token);
                }else if(token_count == 1){
                    n_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
                if(token_count == 0){
                    Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    Vertices[count - 4][Y] = atof(token);
                }else{
                    Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            } else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;

                if(token_count == 1){
                    Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    Faces[i][Y] = atoi(token);
                    //printf("->>>>  %d\n",Faces[count - 144][X]);
                }else if(token_count == 3){
                    Faces[i][Z] = atoi(token);
                    for ( j=0; j < 3; j++ ) {
                        Box[i][0][j] = Vertices[ Faces[i][0] ][j];
                        Box[i][1][j] = Vertices[ Faces[i][0] ][j];
                  }

               for ( k=1; k < 3; k++ )
               for ( j=0; j < 3; j++ ) {
                  w = Vertices[ Faces[i][k] ][j];
                  //printf("->>>>  %d\n",Faces[i][k]);
                  if ( w < Box[i][0][j] ) Box[i][0][j] = w;
                  if ( w > Box[i][1][j] ) Box[i][1][j] = w;
               }
               /*
               printf("Bounding box: (%d,%d,%d);(%d,%d,%d)\n",
                  Box[i][0][0],
                  Box[i][0][1],
                  Box[i][0][2],
                  Box[i][1][0],
                  Box[i][1][1],
                  Box[i][1][2] );
                */
                }
                token_count++;
            }
            token = strtok(NULL," ");
            //free(token);
        }
    }
    if (line)
        free(line);
}
void read_com(void)
{
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    int count = 0;
    float a,b,c;
    fp = fopen("t.off", "r");
    int i ;
    if (fp == NULL)
        exit(EXIT_FAILURE);
    while ((read = getline(&line, &len, fp)) != -1) {
        count++;
        char *token = strtok(line, " ");
        int token_count = 0;
        while (token != NULL) {
            // init facets and vertices
            if(count <= 2){
                printf("setting of file  %s\n", token);
                if(token_count == 0){
                    com_vertices = atoi(token);
                }else if(token_count == 1){
                    com_facets = atoi(token);
                }
                token_count++;
            }else if(count > 3 && count <  n_vertices + 4){
               if(token_count == 0){
                    com_Vertices[count - 4][X] = atof(token);
                }else if(token_count == 1){
                    com_Vertices[count - 4][Y] = atof(token);
                }else{
                    com_Vertices[count - 4][Z] = atof(token);
                }
                token_count++;
            }else if(count >= n_vertices + 4){
                i = count - n_vertices - 4;
                if(token_count == 1){
                    com_Faces[i][X] = atoi(token);
                }else if(token_count == 2){
                    com_Faces[i][Y] = atoi(token);
                }else if(token_count == 3){
                    com_Faces[i][Z] = atoi(token);
                }
                token_count++;
            }
            token = strtok(NULL, " ");
        }
    }
    if (line)
        free(line);
}
